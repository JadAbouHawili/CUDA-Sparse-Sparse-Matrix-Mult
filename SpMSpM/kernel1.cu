#include "hip/hip_runtime.h"

#include "common.h"

// assigning a threadblock per row
#include "common.h"
#include <iostream>
// #include <unordered_map>

#define BLOCK_DIM 256
// #define temp_size (750 / 2)

using namespace std;

__global__ void mul_kernel_opt(CSRMatrix *csrMatrix1, CSRMatrix *csrMatrix2,
                               COOMatrix *cooMatrix3) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int row = i / BLOCK_DIM;
  int num_nonzeros_row =
      csrMatrix1->rowPtrs[row + 1] - csrMatrix1->rowPtrs[row];

  extern __shared__ float temp[];
  //__shared__ float temp[temp_size - 1];
  //__shared__ int temp_indices[temp_size - 1];
  //__shared__ int incr = 0;
  if (threadIdx.x == 0) {
    for (int k = 0; k < csrMatrix2->numCols; k++) {
      temp[k] = 0.0;
    }
  }
  __syncthreads();
  // if (threadIdx.x < num_nonzeros_row) {
  for (int iter = threadIdx.x; iter < num_nonzeros_row; iter += BLOCK_DIM) {

    // iterate over every row in matrix 2

    int col = csrMatrix1->colIdxs[csrMatrix1->rowPtrs[row] + iter];
    float val = csrMatrix1->values[csrMatrix1->rowPtrs[row] + iter];

    int row_start_2 = csrMatrix2->rowPtrs[col];
    int row_end_2 = csrMatrix2->rowPtrs[col + 1];

    for (int k = row_start_2; k < row_end_2; k++) {
      int col2 = csrMatrix2->colIdxs[k];
      float val2 = csrMatrix2->values[k];

      float store = val * val2;
      // col 2 should be indexed on consecutive locations and then when
      // looping over it accessing another array with the col indices (more
      // shared memory?)
      //
      // might lead to illegal memory access
      // int index = atomicAdd(&incr, 1);
      // temp_indices[index] = col2;
      atomicAdd(&temp[col2], store);
      //
    }
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    for (int k = 0; k < csrMatrix2->numCols; k++) {
      if (temp[k] != 0) {
        int index = atomicAdd(&cooMatrix3->numNonzeros, 1);
        cooMatrix3->rowIdxs[index] = row;
        // must change this
        cooMatrix3->colIdxs[index] = k;
        cooMatrix3->values[index] = temp[k];
      }
    }
  }
}

void spmspm_gpu1(COOMatrix *cooMatrix1, CSRMatrix *csrMatrix1,
                 CSCMatrix *cscMatrix1, COOMatrix *cooMatrix2,
                 CSRMatrix *csrMatrix2, CSCMatrix *cscMatrix2,
                 COOMatrix *cooMatrix3, unsigned int numRows1,
                 unsigned int numRows2, unsigned int numCols2,
                 unsigned int numNonzeros1, unsigned int numNonzeros2) {

  // CSR CSR

  int threadsPerBlock = BLOCK_DIM;
  int num_Blocks = numRows1;

  mul_kernel_opt<<<num_Blocks, threadsPerBlock, numCols2>>>(
      csrMatrix1, csrMatrix2, cooMatrix3);
}
